#include <hip/hip_runtime.h>
#include <cstdio>

//extern "C" {
//    #include "fwi_propagator.h"
//}


#define C0 1.2f
#define C1 1.4f
#define C2 1.6f
#define C3 1.8f


__device__ inline
int IDX (const int z, 
         const int x, 
         const int y, 
         const int dimmz, 
         const int dimmx)
{
    return (y*dimmx*dimmz) + (x*dimmz) + (z);
};

__device__ inline
float stencil_Z (const int off,
                 const float* __restrict__ ptr,
                 const float    dzi,
                 const int z,
                 const int x,
                 const int y,
                 const int dimmz,
                 const int dimmx)
{
    return  ((C0 * ( ptr[IDX(z  +off,x,y,dimmz,dimmx)] - ptr[IDX(z-1+off,x,y,dimmz,dimmx)]) +
              C1 * ( ptr[IDX(z+1+off,x,y,dimmz,dimmx)] - ptr[IDX(z-2+off,x,y,dimmz,dimmx)]) +
              C2 * ( ptr[IDX(z+2+off,x,y,dimmz,dimmx)] - ptr[IDX(z-3+off,x,y,dimmz,dimmx)]) +
              C3 * ( ptr[IDX(z+3+off,x,y,dimmz,dimmx)] - ptr[IDX(z-4+off,x,y,dimmz,dimmx)])) * dzi );
};

__device__ inline
float stencil_X(const int off,
                const float* __restrict__ ptr,
                const float dxi,
                const int z,
                const int x,
                const int y,
                const int dimmz,
                const int dimmx)
{
    return ((C0 * ( ptr[IDX(z,x  +off,y,dimmz,dimmx)] - ptr[IDX(z,x-1+off,y,dimmz,dimmx)]) +
             C1 * ( ptr[IDX(z,x+1+off,y,dimmz,dimmx)] - ptr[IDX(z,x-2+off,y,dimmz,dimmx)]) +
             C2 * ( ptr[IDX(z,x+2+off,y,dimmz,dimmx)] - ptr[IDX(z,x-3+off,y,dimmz,dimmx)]) +
             C3 * ( ptr[IDX(z,x+3+off,y,dimmz,dimmx)] - ptr[IDX(z,x-4+off,y,dimmz,dimmx)])) * dxi );
};

__device__ inline
float stencil_Y(const int off,
                const float* __restrict__ ptr,
                const float dyi,
                const int z,
                const int x,
                const int y,
                const int dimmz,
                const int dimmx)
{
    return ((C0 * ( ptr[IDX(z,x,y  +off,dimmz,dimmx)] - ptr[IDX(z,x,y-1+off,dimmz,dimmx)]) +
             C1 * ( ptr[IDX(z,x,y+1+off,dimmz,dimmx)] - ptr[IDX(z,x,y-2+off,dimmz,dimmx)]) +
             C2 * ( ptr[IDX(z,x,y+2+off,dimmz,dimmx)] - ptr[IDX(z,x,y-3+off,dimmz,dimmx)]) +
             C3 * ( ptr[IDX(z,x,y+3+off,dimmz,dimmx)] - ptr[IDX(z,x,y-4+off,dimmz,dimmx)])) * dyi );
};

/* -------------------------------------------------------------------- */
/*                     KERNELS FOR VELOCITY                             */
/* -------------------------------------------------------------------- */

__device__ inline
float rho_BL (const float* __restrict__ rho,
              const int z,
              const int x,
              const int y,
              const int dimmz,
              const int dimmx)
{
    return (2.0f / (rho[IDX(z,x,y,dimmz,dimmx)] + rho[IDX(z+1,x,y,dimmz,dimmx)]));
};

__device__ inline
float rho_TR (const float* __restrict__ rho,
              const int z,
              const int x,
              const int y,
              const int dimmz,
              const int dimmx)
{
    return (2.0f / (rho[IDX(z,x,y,dimmz,dimmx)] + rho[IDX(z,x+1,y,dimmz,dimmx)]));
};

__device__ inline
float rho_BR (const float* __restrict__ rho,
              const int z,
              const int x,
              const int y,
              const int dimmz,
              const int dimmx)
{
    return ( 8.0f/ ( rho[IDX(z  ,x  ,y  ,dimmz,dimmx)] +
                     rho[IDX(z+1,x  ,y  ,dimmz,dimmx)] +
                     rho[IDX(z  ,x+1,y  ,dimmz,dimmx)] +
                     rho[IDX(z  ,x  ,y+1,dimmz,dimmx)] +
                     rho[IDX(z  ,x+1,y+1,dimmz,dimmx)] +
                     rho[IDX(z+1,x+1,y  ,dimmz,dimmx)] +
                     rho[IDX(z+1,x  ,y+1,dimmz,dimmx)] +
                     rho[IDX(z+1,x+1,y+1,dimmz,dimmx)]) );
};

__device__ inline
float rho_TL (const float* __restrict__ rho,
              const int z,
              const int x,
              const int y,
              const int dimmz,
              const int dimmx)
{
    return (2.0f / (rho[IDX(z,x,y,dimmz,dimmx)] + rho[IDX(z,x,y+1,dimmz,dimmx)]));
};

#ifdef OPTIMIZED
template <const int HALO = 4, 
          const int BLOCK_DIM_X = 16,
          const int BLOCK_DIM_Y = 16>
__global__
__launch_bounds__(128, 16) 
void compute_component_vcell_TL_cuda_k ( float* __restrict__ vptr,
                                   const float* __restrict__ szptr,
                                   const float* __restrict__ sxptr,
                                   const float* __restrict__ syptr,
                                   const float* __restrict__ rho,
                                   const float           dt,
                                   const float           dzi,
                                   const float           dxi,
                                   const float           dyi,
                                   const int             nz0,
                                   const int             nzf,
                                   const int             nx0,
                                   const int             nxf,
                                   const int             ny0,
                                   const int             nyf,
                                   const int             SZ,
                                   const int             SX,
                                   const int             SY,
                                   const int             dimmz,
                                   const int             dimmx)
{
    for(int z = blockIdx.x * blockDim.x + threadIdx.x + nz0; 
            z < nzf; 
            z += gridDim.x * blockDim.x)
    {
        for(int x = blockIdx.y * blockDim.y + threadIdx.y + nx0; 
                x < nxf; 
                x += gridDim.y * blockDim.y)
        {
            __shared__ float sx_smem[BLOCK_DIM_Y][BLOCK_DIM_X+2*HALO];
            float sy_front1, sy_front2, sy_front3, sy_front4;
            float sy_back1, sy_back2, sy_back3, sy_back4;
            float sy_current;

            sy_back3   = syptr[IDX(z,x,ny0-HALO+0+SY,dimmz,dimmx)];
            sy_back2   = syptr[IDX(z,x,ny0-HALO+1+SY,dimmz,dimmx)];
            sy_back1   = syptr[IDX(z,x,ny0-HALO+2+SY,dimmz,dimmx)];
            sy_current = syptr[IDX(z,x,ny0-HALO+3+SY,dimmz,dimmx)];
            sy_front1  = syptr[IDX(z,x,ny0-HALO+4+SY,dimmz,dimmx)];
            sy_front2  = syptr[IDX(z,x,ny0-HALO+5+SY,dimmz,dimmx)];
            sy_front3  = syptr[IDX(z,x,ny0-HALO+6+SY,dimmz,dimmx)];
            sy_front4  = syptr[IDX(z,x,ny0-HALO+7+SY,dimmz,dimmx)];

            float rho_current, rho_front1;
            rho_front1 = rho[IDX(z,x,ny0,dimmz,dimmx)];

            for(int y = ny0; 
                    y < nyf; 
                    y++)
            {
                /////////// register tiling-advance plane ////////////////
                sy_back4   = sy_back3;
                sy_back3   = sy_back2;
                sy_back2   = sy_back1;
                sy_back1   = sy_current;
                sy_current = sy_front1;
                sy_front1  = sy_front2;
                sy_front2  = sy_front3;
                sy_front3  = sy_front4;
                sy_front4  = syptr[IDX(z,x,y+SY+HALO,dimmz,dimmx)];
                ///////////////////////
                rho_current = rho_front1;
                rho_front1  = rho[IDX(z,x,y+1,dimmz,dimmx)];
                //////////////////////////////////////////////////////////

                const int tx = threadIdx.x+HALO;
                const int ty = threadIdx.y;
                ///////////// intra-block communication///////////////////
                sx_smem[ty][tx] = sxptr[IDX(z,x+SX,y,dimmz,dimmx)];
                //if (threadIdx.y < HALO)
                //{
                //    sx_smem[threadIdx.y                 ][tx] = sxptr[IDX(z,x+SX-HALO,       y,dimmz,dimmx)];
                //    sx_smem[threadIdx.y+BLOCK_DIM_Y+HALO][tx] = sxptr[IDX(z,x+SX+BLOCK_DIM_X,y,dimmz,dimmx)];
                //}
                if (threadIdx.x < HALO)
                {
                    sx_smem[ty][threadIdx.x                 ] = sxptr[IDX(z,x+SX-HALO,       y,dimmz,dimmx)];
                    sx_smem[ty][threadIdx.x+BLOCK_DIM_X+HALO] = sxptr[IDX(z,x+SX+BLOCK_DIM_X,y,dimmz,dimmx)];
                }
                /////////////////////////////////////////////////////////
                
                ///////////// intra-warp communication /////////////////
                float sz_current = szptr[IDX(z+SZ,x,y,dimmz,dimmx)];
                float sz_front3 = __shfl_up(sz_current, 3);
                float sz_front2 = __shfl_up(sz_current, 2);
                float sz_front1 = __shfl_up(sz_current, 1);
                float sz_back1  = __shfl_down(sz_current, 1);
                float sz_back2  = __shfl_down(sz_current, 2);
                float sz_back3  = __shfl_down(sz_current, 3);
                float sz_back4  = __shfl_down(sz_current, 4);
                ////////////////////////////////////////////////////////
                __syncthreads();

                const float lrho = (2.0f / (rho_current + rho_front1));

                const float stz = ((C0 * ( sz_current - sz_back1 ) +
                                    C1 * ( sz_front1  - sz_back2 ) +
                                    C2 * ( sz_front2  - sz_back3 ) +
                                    C3 * ( sz_front3  - sz_back4 )) * dzi );
                
                const float stx = ((C0 * ( sx_smem[ty][tx  ] - sx_smem[ty][tx-1] ) +
                                    C1 * ( sx_smem[ty][tx+1] - sx_smem[ty][tx-2] ) +
                                    C2 * ( sx_smem[ty][tx+2] - sx_smem[ty][tx-3] ) +
                                    C3 * ( sx_smem[ty][tx+3] - sx_smem[ty][tx-4] )) * dxi );

                const float sty = ((C0 * ( sy_current - sy_back1 ) +
                                    C1 * ( sy_front1  - sy_back2 ) +
                                    C2 * ( sy_front2  - sy_back3 ) +
                                    C3 * ( sy_front3  - sy_back4 )) * dyi );

                vptr[IDX(z,x,y,dimmz,dimmx)] += (stx  + sty  + stz) * dt * lrho;
            }
        }
    }
}
#else
__global__
void compute_component_vcell_TL_cuda_k ( float* __restrict__ vptr,
                                   const float* __restrict__ szptr,
                                   const float* __restrict__ sxptr,
                                   const float* __restrict__ syptr,
                                   const float* __restrict__ rho,
                                   const float           dt,
                                   const float           dzi,
                                   const float           dxi,
                                   const float           dyi,
                                   const int             nz0,
                                   const int             nzf,
                                   const int             nx0,
                                   const int             nxf,
                                   const int             ny0,
                                   const int             nyf,
                                   const int             SZ,
                                   const int             SX,
                                   const int             SY,
                                   const int             dimmz,
                                   const int             dimmx)
{
    for(int z = blockIdx.x * blockDim.x + threadIdx.x + nz0; 
            z < nzf; 
            z += gridDim.x * blockDim.x)
    {
        for(int x = blockIdx.y * blockDim.y + threadIdx.y + nx0; 
                x < nxf; 
                x += gridDim.y * blockDim.y)
        {
            for(int y = ny0; 
                    y < nyf; 
                    y++)
            {
                const float lrho = rho_TL(rho, z, x, y, dimmz, dimmx);
                
                const float stx  = stencil_X( SX, sxptr, dxi, z, x, y, dimmz, dimmx);
                const float sty  = stencil_Y( SY, syptr, dyi, z, x, y, dimmz, dimmx);
                const float stz  = stencil_Z( SZ, szptr, dzi, z, x, y, dimmz, dimmx);
                
                vptr[IDX(z,x,y,dimmz,dimmx)] += (stx  + sty  + stz) * dt * lrho;
            }
        }
    }
}
#endif

extern "C"
void compute_component_vcell_TL_cuda ( float* vptr,
                                 const float* szptr,
                                 const float* sxptr,
                                 const float* syptr,
                                 const float* rho,
                                 const float  dt,
                                 const float  dzi,
                                 const float  dxi,
                                 const float  dyi,
                                 const int    nz0,
                                 const int    nzf,
                                 const int    nx0,
                                 const int    nxf,
                                 const int    ny0,
                                 const int    nyf,
                                 const int    SZ,
                                 const int    SX,
                                 const int    SY,
                                 const int    dimmz,
                                 const int    dimmx,
                                 void*        stream)
{
    const int block_dim_x = 32;
    const int block_dim_y = 4;


    dim3 grid_dim( ((nzf-nz0) + block_dim_x-1)/block_dim_x,
                   ((nxf-nx0) + block_dim_y-1)/block_dim_y,
                      1 );
    dim3 block_dim(block_dim_x, block_dim_y, 1);

    hipStream_t s = (hipStream_t) stream;

#ifdef OPTIMIZED
    compute_component_vcell_TL_cuda_k<4,block_dim_x,block_dim_y><<<grid_dim, block_dim, 0, s>>>
        (vptr, szptr, sxptr, syptr, rho, dt, dzi, dxi, dyi, 
         nz0, nzf, nx0, nxf, ny0, nyf, SZ, SX, SY, dimmz, dimmx);
#else
    compute_component_vcell_TL_cuda_k<<<grid_dim, block_dim, 0, s>>>
        (vptr, szptr, sxptr, syptr, rho, dt, dzi, dxi, dyi, 
         nz0, nzf, nx0, nxf, ny0, nyf, SZ, SX, SY, dimmz, dimmx);
#endif
};


#ifdef OPTIMIZED
template <const int HALO = 4, 
          const int BLOCK_DIM_X = 16,
          const int BLOCK_DIM_Y = 16>
__global__
__launch_bounds__(128, 16) 
void compute_component_vcell_TR_cuda_k ( float* __restrict__ vptr,
                                   const float* __restrict__ szptr,
                                   const float* __restrict__ sxptr,
                                   const float* __restrict__ syptr,
                                   const float* __restrict__ rho,
                                   const float           dt,
                                   const float           dzi,
                                   const float           dxi,
                                   const float           dyi,
                                   const int             nz0,
                                   const int             nzf,
                                   const int             nx0,
                                   const int             nxf,
                                   const int             ny0,
                                   const int             nyf,
                                   const int             SZ,
                                   const int             SX,
                                   const int             SY,
                                   const int             dimmz,
                                   const int             dimmx)
{
    for(int z = blockIdx.x * blockDim.x + threadIdx.x + nz0; 
            z < nzf; 
            z += gridDim.x * blockDim.x)
    {
        for(int x = blockIdx.y * blockDim.y + threadIdx.y + nx0; 
                x < nxf; 
                x += gridDim.y * blockDim.y)
        {
            __shared__ float sx_smem[BLOCK_DIM_Y][BLOCK_DIM_X+2*HALO];
            float sy_front1, sy_front2, sy_front3, sy_front4;
            float sy_back1, sy_back2, sy_back3, sy_back4;
            float sy_current;

            sy_back3   = syptr[IDX(z,x,ny0-HALO+0+SY,dimmz,dimmx)];
            sy_back2   = syptr[IDX(z,x,ny0-HALO+1+SY,dimmz,dimmx)];
            sy_back1   = syptr[IDX(z,x,ny0-HALO+2+SY,dimmz,dimmx)];
            sy_current = syptr[IDX(z,x,ny0-HALO+3+SY,dimmz,dimmx)];
            sy_front1  = syptr[IDX(z,x,ny0-HALO+4+SY,dimmz,dimmx)];
            sy_front2  = syptr[IDX(z,x,ny0-HALO+5+SY,dimmz,dimmx)];
            sy_front3  = syptr[IDX(z,x,ny0-HALO+6+SY,dimmz,dimmx)];
            sy_front4  = syptr[IDX(z,x,ny0-HALO+7+SY,dimmz,dimmx)];

            __shared__ float rho_smem[BLOCK_DIM_Y+1][BLOCK_DIM_X];

            for(int y = ny0; 
                    y < nyf; 
                    y++)
            {
                /////////// register tiling-advance plane ////////////////
                sy_back4   = sy_back3;
                sy_back3   = sy_back2;
                sy_back2   = sy_back1;
                sy_back1   = sy_current;
                sy_current = sy_front1;
                sy_front1  = sy_front2;
                sy_front2  = sy_front3;
                sy_front3  = sy_front4;
                sy_front4  = syptr[IDX(z,x,y+SY+HALO,dimmz,dimmx)];
                //////////////////////////////////////////////////////////

                const int tx = threadIdx.x+HALO;
                const int ty = threadIdx.y;
                ///////////// intra-block communication///////////////////
                sx_smem[ty][tx] = sxptr[IDX(z,x+SX,y,dimmz,dimmx)];
                //if (threadIdx.y < HALO)
                //{
                //    sx_smem[threadIdx.y                 ][tx] = sxptr[IDX(z,x+SX-HALO,       y,dimmz,dimmx)];
                //    sx_smem[threadIdx.y+BLOCK_DIM_Y+HALO][tx] = sxptr[IDX(z,x+SX+BLOCK_DIM_X,y,dimmz,dimmx)];
                //}
                if (threadIdx.x < HALO)
                {
                    sx_smem[ty][threadIdx.x                 ] = sxptr[IDX(z,x+SX-HALO,       y,dimmz,dimmx)];
                    sx_smem[ty][threadIdx.x+BLOCK_DIM_X+HALO] = sxptr[IDX(z,x+SX+BLOCK_DIM_X,y,dimmz,dimmx)];
                }
                /////////////////////////////////////////////////////////
                rho_smem[ty][threadIdx.x] = rho[IDX(z,x,y,dimmz,dimmx)];
                if (threadIdx.y < 1)
                {
                    rho_smem[ty+BLOCK_DIM_Y+1][threadIdx.x] = rho[IDX(z,x+1,y,dimmz,dimmx)];
                }
                /////////////////////////////////////////////////////////
                ///////////// intra-warp communication /////////////////
                float sz_current = szptr[IDX(z+SZ,x,y,dimmz,dimmx)];
                float sz_front3 = __shfl_up(sz_current, 3);
                float sz_front2 = __shfl_up(sz_current, 2);
                float sz_front1 = __shfl_up(sz_current, 1);
                float sz_back1  = __shfl_down(sz_current, 1);
                float sz_back2  = __shfl_down(sz_current, 2);
                float sz_back3  = __shfl_down(sz_current, 3);
                float sz_back4  = __shfl_down(sz_current, 4);
                ////////////////////////////////////////////////////////
                __syncthreads();

                const float lrho = (2.0f / (rho_smem[ty][threadIdx.x] + rho_smem[ty+1][threadIdx.x]));

                const float stz = ((C0 * ( sz_current - sz_back1 ) +
                                    C1 * ( sz_front1  - sz_back2 ) +
                                    C2 * ( sz_front2  - sz_back3 ) +
                                    C3 * ( sz_front3  - sz_back4 )) * dzi );
                
                const float stx = ((C0 * ( sx_smem[ty][tx  ] - sx_smem[ty][tx-1] ) +
                                    C1 * ( sx_smem[ty][tx+1] - sx_smem[ty][tx-2] ) +
                                    C2 * ( sx_smem[ty][tx+2] - sx_smem[ty][tx-3] ) +
                                    C3 * ( sx_smem[ty][tx+3] - sx_smem[ty][tx-4] )) * dxi );

                const float sty = ((C0 * ( sy_current - sy_back1 ) +
                                    C1 * ( sy_front1  - sy_back2 ) +
                                    C2 * ( sy_front2  - sy_back3 ) +
                                    C3 * ( sy_front3  - sy_back4 )) * dyi );

                vptr[IDX(z,x,y,dimmz,dimmx)] += (stx  + sty  + stz) * dt * lrho;
            }
        }
    }
}
#else
__global__
void compute_component_vcell_TR_cuda_k ( float* __restrict__ vptr,
                                   const float* __restrict__ szptr,
                                   const float* __restrict__ sxptr,
                                   const float* __restrict__ syptr,
                                   const float* __restrict__ rho,
                                   const float           dt,
                                   const float           dzi,
                                   const float           dxi,
                                   const float           dyi,
                                   const int             nz0,
                                   const int             nzf,
                                   const int             nx0,
                                   const int             nxf,
                                   const int             ny0,
                                   const int             nyf,
                                   const int             SZ,
                                   const int             SX,
                                   const int             SY,
                                   const int             dimmz,
                                   const int             dimmx)
{
    for(int z = blockIdx.x * blockDim.x + threadIdx.x + nz0; 
            z < nzf; 
            z += gridDim.x * blockDim.x)
    {
        for(int x = blockIdx.y * blockDim.y + threadIdx.y + nx0; 
                x < nxf; 
                x += gridDim.y * blockDim.y)
        {
            for(int y = ny0; 
                    y < nyf; 
                    y++)
            {
                const float lrho = rho_TR(rho, z, x, y, dimmz, dimmx);
                
                const float stx  = stencil_X( SX, sxptr, dxi, z, x, y, dimmz, dimmx);
                const float sty  = stencil_Y( SY, syptr, dyi, z, x, y, dimmz, dimmx);
                const float stz  = stencil_Z( SZ, szptr, dzi, z, x, y, dimmz, dimmx);
                
                vptr[IDX(z,x,y,dimmz,dimmx)] += (stx  + sty  + stz) * dt * lrho;
            }
        }
    }
}
#endif

extern "C"
void compute_component_vcell_TR_cuda ( float* vptr,
                                 const float* szptr,
                                 const float* sxptr,
                                 const float* syptr,
                                 const float* rho,
                                 const float  dt,
                                 const float  dzi,
                                 const float  dxi,
                                 const float  dyi,
                                 const int    nz0,
                                 const int    nzf,
                                 const int    nx0,
                                 const int    nxf,
                                 const int    ny0,
                                 const int    nyf,
                                 const int    SZ,
                                 const int    SX,
                                 const int    SY,
                                 const int    dimmz,
                                 const int    dimmx,
                                 void*        stream)
{
    const int block_dim_x = 32;
    const int block_dim_y = 4;


    dim3 grid_dim( ((nzf-nz0) + block_dim_x-1)/block_dim_x,
                   ((nxf-nx0) + block_dim_y-1)/block_dim_y,
                      1 );
    dim3 block_dim(block_dim_x, block_dim_y, 1);

    hipStream_t s = (hipStream_t) stream;

#ifdef OPTIMIZED
    compute_component_vcell_TR_cuda_k<4,block_dim_x,block_dim_y><<<grid_dim, block_dim, 0, s>>>
        (vptr, szptr, sxptr, syptr, rho, dt, dzi, dxi, dyi, 
         nz0, nzf, nx0, nxf, ny0, nyf, SZ, SX, SY, dimmz, dimmx);
#else
    compute_component_vcell_TR_cuda_k<<<grid_dim, block_dim, 0, s>>>
        (vptr, szptr, sxptr, syptr, rho, dt, dzi, dxi, dyi, 
         nz0, nzf, nx0, nxf, ny0, nyf, SZ, SX, SY, dimmz, dimmx);
#endif
};


#ifdef OPTIMIZED
template <const int HALO = 4, 
          const int BLOCK_DIM_X = 16,
          const int BLOCK_DIM_Y = 16>
__global__
__launch_bounds__(128, 16) 
void compute_component_vcell_BR_cuda_k ( float* __restrict__ vptr,
                                   const float* __restrict__ szptr,
                                   const float* __restrict__ sxptr,
                                   const float* __restrict__ syptr,
                                   const float* __restrict__ rho,
                                   const float           dt,
                                   const float           dzi,
                                   const float           dxi,
                                   const float           dyi,
                                   const int             nz0,
                                   const int             nzf,
                                   const int             nx0,
                                   const int             nxf,
                                   const int             ny0,
                                   const int             nyf,
                                   const int             SZ,
                                   const int             SX,
                                   const int             SY,
                                   const int             dimmz,
                                   const int             dimmx)
{
    for(int z = blockIdx.x * blockDim.x + threadIdx.x + nz0; 
            z < nzf; 
            z += gridDim.x * blockDim.x)
    {
        for(int x = blockIdx.y * blockDim.y + threadIdx.y + nx0; 
                x < nxf; 
                x += gridDim.y * blockDim.y)
        {
            __shared__ float sx_smem[BLOCK_DIM_Y][BLOCK_DIM_X+2*HALO];
            float sy_front1, sy_front2, sy_front3, sy_front4;
            float sy_back1, sy_back2, sy_back3, sy_back4;
            float sy_current;

            sy_back3   = syptr[IDX(z,x,ny0-HALO+0+SY,dimmz,dimmx)];
            sy_back2   = syptr[IDX(z,x,ny0-HALO+1+SY,dimmz,dimmx)];
            sy_back1   = syptr[IDX(z,x,ny0-HALO+2+SY,dimmz,dimmx)];
            sy_current = syptr[IDX(z,x,ny0-HALO+3+SY,dimmz,dimmx)];
            sy_front1  = syptr[IDX(z,x,ny0-HALO+4+SY,dimmz,dimmx)];
            sy_front2  = syptr[IDX(z,x,ny0-HALO+5+SY,dimmz,dimmx)];
            sy_front3  = syptr[IDX(z,x,ny0-HALO+6+SY,dimmz,dimmx)];
            sy_front4  = syptr[IDX(z,x,ny0-HALO+7+SY,dimmz,dimmx)];

            __shared__ float rho_smem[BLOCK_DIM_Y+1][BLOCK_DIM_X+1];
            float rho_current, rho_front1;
            rho_front1 = rho[IDX(z,x,ny0,dimmz,dimmx)];

            for(int y = ny0; 
                    y < nyf; 
                    y++)
            {
                /////////// register tiling-advance plane ////////////////
                sy_back4   = sy_back3;
                sy_back3   = sy_back2;
                sy_back2   = sy_back1;
                sy_back1   = sy_current;
                sy_current = sy_front1;
                sy_front1  = sy_front2;
                sy_front2  = sy_front3;
                sy_front3  = sy_front4;
                sy_front4  = syptr[IDX(z,x,y+SY+HALO,dimmz,dimmx)];
                ///////////////////////
                rho_current = rho_front1;
                rho_front1  = rho[IDX(z,x,y+1,dimmz,dimmx)];
                //////////////////////////////////////////////////////////

                const int tx = threadIdx.x+HALO;
                const int ty = threadIdx.y;
                ///////////// intra-block communication///////////////////
                sx_smem[ty][tx] = sxptr[IDX(z,x+SX,y,dimmz,dimmx)];
                //if (threadIdx.y < HALO)
                //{
                //    sx_smem[threadIdx.y                 ][tx] = sxptr[IDX(z,x+SX-HALO,       y,dimmz,dimmx)];
                //    sx_smem[threadIdx.y+BLOCK_DIM_Y+HALO][tx] = sxptr[IDX(z,x+SX+BLOCK_DIM_X,y,dimmz,dimmx)];
                //}
                if (threadIdx.x < HALO)
                {
                    sx_smem[ty][threadIdx.x                 ] = sxptr[IDX(z,x+SX-HALO,       y,dimmz,dimmx)];
                    sx_smem[ty][threadIdx.x+BLOCK_DIM_X+HALO] = sxptr[IDX(z,x+SX+BLOCK_DIM_X,y,dimmz,dimmx)];
                }
                /////////////////////////////////////////////////////////
                rho_smem[ty][threadIdx.x] = rho[IDX(z,x,y,dimmz,dimmx)];
                if (threadIdx.y < 1)
                    rho_smem[ty+BLOCK_DIM_Y+1][threadIdx.x              ] = rho[IDX(z,x+BLOCK_DIM_Y+1,y,dimmz,dimmx)];
                if (threadIdx.x < 1)
                    rho_smem[ty              ][threadIdx.x+BLOCK_DIM_X+1] = rho[IDX(z+BLOCK_DIM_X+1,x,y,dimmz,dimmx)];
                /////////////////////////////////////////////////////////
                
                ///////////// intra-warp communication /////////////////
                float sz_current = szptr[IDX(z+SZ,x,y,dimmz,dimmx)];
                float sz_front3 = __shfl_up(sz_current, 3);
                float sz_front2 = __shfl_up(sz_current, 2);
                float sz_front1 = __shfl_up(sz_current, 1);
                float sz_back1  = __shfl_down(sz_current, 1);
                float sz_back2  = __shfl_down(sz_current, 2);
                float sz_back3  = __shfl_down(sz_current, 3);
                float sz_back4  = __shfl_down(sz_current, 4);
                ////////////////////////////////////////////////////////
                __syncthreads();

                const float lrho = (8.0f/ ( rho_current                       +
                                            rho_smem[ty  ][threadIdx.x+1]     +
                                            rho_smem[ty+1][threadIdx.x  ]     +
                                            rho_front1                        +
                                            rho[IDX(z  ,x+1,y+1,dimmz,dimmx)] +
                                            rho[IDX(z+1,x+1,y  ,dimmz,dimmx)] +
                                            rho[IDX(z+1,x  ,y+1,dimmz,dimmx)] +
                                            rho[IDX(z+1,x+1,y+1,dimmz,dimmx)]) );

                const float stz = ((C0 * ( sz_current - sz_back1 ) +
                                    C1 * ( sz_front1  - sz_back2 ) +
                                    C2 * ( sz_front2  - sz_back3 ) +
                                    C3 * ( sz_front3  - sz_back4 )) * dzi );
                
                const float stx = ((C0 * ( sx_smem[ty][tx  ] - sx_smem[ty][tx-1] ) +
                                    C1 * ( sx_smem[ty][tx+1] - sx_smem[ty][tx-2] ) +
                                    C2 * ( sx_smem[ty][tx+2] - sx_smem[ty][tx-3] ) +
                                    C3 * ( sx_smem[ty][tx+3] - sx_smem[ty][tx-4] )) * dxi );

                const float sty = ((C0 * ( sy_current - sy_back1 ) +
                                    C1 * ( sy_front1  - sy_back2 ) +
                                    C2 * ( sy_front2  - sy_back3 ) +
                                    C3 * ( sy_front3  - sy_back4 )) * dyi );

                vptr[IDX(z,x,y,dimmz,dimmx)] += (stx  + sty  + stz) * dt * lrho;
            }
        }
    }
}
#else
__global__
void compute_component_vcell_BR_cuda_k ( float* __restrict__ vptr,
                                   const float* __restrict__ szptr,
                                   const float* __restrict__ sxptr,
                                   const float* __restrict__ syptr,
                                   const float* __restrict__ rho,
                                   const float           dt,
                                   const float           dzi,
                                   const float           dxi,
                                   const float           dyi,
                                   const int             nz0,
                                   const int             nzf,
                                   const int             nx0,
                                   const int             nxf,
                                   const int             ny0,
                                   const int             nyf,
                                   const int             SZ,
                                   const int             SX,
                                   const int             SY,
                                   const int             dimmz,
                                   const int             dimmx)
{
    for(int z = blockIdx.x * blockDim.x + threadIdx.x + nz0; 
            z < nzf; 
            z += gridDim.x * blockDim.x)
    {
        for(int x = blockIdx.y * blockDim.y + threadIdx.y + nx0; 
                x < nxf; 
                x += gridDim.y * blockDim.y)
        {
            for(int y = ny0; 
                    y < nyf; 
                    y++)
            {
                const float lrho = rho_BR(rho, z, x, y, dimmz, dimmx);
                
                const float stx  = stencil_X( SX, sxptr, dxi, z, x, y, dimmz, dimmx);
                const float sty  = stencil_Y( SY, syptr, dyi, z, x, y, dimmz, dimmx);
                const float stz  = stencil_Z( SZ, szptr, dzi, z, x, y, dimmz, dimmx);
                
                vptr[IDX(z,x,y,dimmz,dimmx)] += (stx  + sty  + stz) * dt * lrho;
            }
        }
    }
}
#endif

extern "C"
void compute_component_vcell_BR_cuda ( float* vptr,
                                 const float* szptr,
                                 const float* sxptr,
                                 const float* syptr,
                                 const float* rho,
                                 const float  dt,
                                 const float  dzi,
                                 const float  dxi,
                                 const float  dyi,
                                 const int    nz0,
                                 const int    nzf,
                                 const int    nx0,
                                 const int    nxf,
                                 const int    ny0,
                                 const int    nyf,
                                 const int    SZ,
                                 const int    SX,
                                 const int    SY,
                                 const int    dimmz,
                                 const int    dimmx,
                                 void*        stream)
{
    const int block_dim_x = 32;
    const int block_dim_y = 4;


    dim3 grid_dim( ((nzf-nz0) + block_dim_x-1)/block_dim_x,
                   ((nxf-nx0) + block_dim_y-1)/block_dim_y,
                      1 );
    dim3 block_dim(block_dim_x, block_dim_y, 1);

    hipStream_t s = (hipStream_t) stream;

#ifdef OPTIMIZED
    compute_component_vcell_BR_cuda_k<4,block_dim_x,block_dim_y><<<grid_dim, block_dim, 0, s>>>
        (vptr, szptr, sxptr, syptr, rho, dt, dzi, dxi, dyi, 
         nz0, nzf, nx0, nxf, ny0, nyf, SZ, SX, SY, dimmz, dimmx);
#else
    compute_component_vcell_BR_cuda_k<<<grid_dim, block_dim, 0, s>>>
        (vptr, szptr, sxptr, syptr, rho, dt, dzi, dxi, dyi, 
         nz0, nzf, nx0, nxf, ny0, nyf, SZ, SX, SY, dimmz, dimmx);
#endif
};

#ifdef OPTIMIZED
template <const int HALO = 4, 
          const int BLOCK_DIM_X = 16,
          const int BLOCK_DIM_Y = 16>
__global__
__launch_bounds__(128, 16) 
void compute_component_vcell_BL_cuda_k ( float* __restrict__ vptr,
                                   const float* __restrict__ szptr,
                                   const float* __restrict__ sxptr,
                                   const float* __restrict__ syptr,
                                   const float* __restrict__ rho,
                                   const float           dt,
                                   const float           dzi,
                                   const float           dxi,
                                   const float           dyi,
                                   const int             nz0,
                                   const int             nzf,
                                   const int             nx0,
                                   const int             nxf,
                                   const int             ny0,
                                   const int             nyf,
                                   const int             SZ,
                                   const int             SX,
                                   const int             SY,
                                   const int             dimmz,
                                   const int             dimmx)
{
    for(int z = blockIdx.x * blockDim.x + threadIdx.x + nz0; 
            z < nzf; 
            z += gridDim.x * blockDim.x)
    {
        for(int x = blockIdx.y * blockDim.y + threadIdx.y + nx0; 
                x < nxf; 
                x += gridDim.y * blockDim.y)
        {
            __shared__ float sx_smem[BLOCK_DIM_Y][BLOCK_DIM_X+2*HALO];
            float sy_front1, sy_front2, sy_front3, sy_front4;
            float sy_back1, sy_back2, sy_back3, sy_back4;
            float sy_current;

            sy_back3   = syptr[IDX(z,x,ny0-HALO+0+SY,dimmz,dimmx)];
            sy_back2   = syptr[IDX(z,x,ny0-HALO+1+SY,dimmz,dimmx)];
            sy_back1   = syptr[IDX(z,x,ny0-HALO+2+SY,dimmz,dimmx)];
            sy_current = syptr[IDX(z,x,ny0-HALO+3+SY,dimmz,dimmx)];
            sy_front1  = syptr[IDX(z,x,ny0-HALO+4+SY,dimmz,dimmx)];
            sy_front2  = syptr[IDX(z,x,ny0-HALO+5+SY,dimmz,dimmx)];
            sy_front3  = syptr[IDX(z,x,ny0-HALO+6+SY,dimmz,dimmx)];
            sy_front4  = syptr[IDX(z,x,ny0-HALO+7+SY,dimmz,dimmx)];

            for(int y = ny0; 
                    y < nyf; 
                    y++)
            {
                /////////// register tiling-advance plane ////////////////
                sy_back4   = sy_back3;
                sy_back3   = sy_back2;
                sy_back2   = sy_back1;
                sy_back1   = sy_current;
                sy_current = sy_front1;
                sy_front1  = sy_front2;
                sy_front2  = sy_front3;
                sy_front3  = sy_front4;
                sy_front4  = syptr[IDX(z,x,y+SY+HALO,dimmz,dimmx)];
                //////////////////////////////////////////////////////////

                const int tx = threadIdx.x+HALO;
                const int ty = threadIdx.y;
                ///////////// intra-block communication///////////////////
                sx_smem[ty][tx] = sxptr[IDX(z,x+SX,y,dimmz,dimmx)];
                //if (threadIdx.y < HALO)
                //{
                //    sx_smem[threadIdx.y                 ][tx] = sxptr[IDX(z,x+SX-HALO,       y,dimmz,dimmx)];
                //    sx_smem[threadIdx.y+BLOCK_DIM_Y+HALO][tx] = sxptr[IDX(z,x+SX+BLOCK_DIM_X,y,dimmz,dimmx)];
                //}
                if (threadIdx.x < HALO)
                {
                    sx_smem[ty][threadIdx.x                 ] = sxptr[IDX(z,x+SX-HALO,       y,dimmz,dimmx)];
                    sx_smem[ty][threadIdx.x+BLOCK_DIM_X+HALO] = sxptr[IDX(z,x+SX+BLOCK_DIM_X,y,dimmz,dimmx)];
                }
                /////////////////////////////////////////////////////////
                
                ///////////// intra-warp communication /////////////////
                float sz_current = szptr[IDX(z+SZ,x,y,dimmz,dimmx)];
                float sz_front3 = __shfl_up(sz_current, 3);
                float sz_front2 = __shfl_up(sz_current, 2);
                float sz_front1 = __shfl_up(sz_current, 1);
                float sz_back1  = __shfl_down(sz_current, 1);
                float sz_back2  = __shfl_down(sz_current, 2);
                float sz_back3  = __shfl_down(sz_current, 3);
                float sz_back4  = __shfl_down(sz_current, 4);
                ////////////////////////////////////////////////////////
                float rho_current  = rho[IDX(z,x,y,dimmz,dimmx)];
                float rho_front1   = __shfl_up(rho_current, 1);
                ////////////////////////////////////////////////////////
                __syncthreads();

                const float lrho = (2.0f / (rho_current + rho_front1));

                const float stz = ((C0 * ( sz_current - sz_back1 ) +
                                    C1 * ( sz_front1  - sz_back2 ) +
                                    C2 * ( sz_front2  - sz_back3 ) +
                                    C3 * ( sz_front3  - sz_back4 )) * dzi );
                
                const float stx = ((C0 * ( sx_smem[ty][tx  ] - sx_smem[ty][tx-1] ) +
                                    C1 * ( sx_smem[ty][tx+1] - sx_smem[ty][tx-2] ) +
                                    C2 * ( sx_smem[ty][tx+2] - sx_smem[ty][tx-3] ) +
                                    C3 * ( sx_smem[ty][tx+3] - sx_smem[ty][tx-4] )) * dxi );

                const float sty = ((C0 * ( sy_current - sy_back1 ) +
                                    C1 * ( sy_front1  - sy_back2 ) +
                                    C2 * ( sy_front2  - sy_back3 ) +
                                    C3 * ( sy_front3  - sy_back4 )) * dyi );

                vptr[IDX(z,x,y,dimmz,dimmx)] += (stx  + sty  + stz) * dt * lrho;
            }
        }
    }
}
#else
__global__
void compute_component_vcell_BL_cuda_k ( float* __restrict__ vptr,
                                   const float* __restrict__ szptr,
                                   const float* __restrict__ sxptr,
                                   const float* __restrict__ syptr,
                                   const float* __restrict__ rho,
                                   const float           dt,
                                   const float           dzi,
                                   const float           dxi,
                                   const float           dyi,
                                   const int             nz0,
                                   const int             nzf,
                                   const int             nx0,
                                   const int             nxf,
                                   const int             ny0,
                                   const int             nyf,
                                   const int             SZ,
                                   const int             SX,
                                   const int             SY,
                                   const int             dimmz,
                                   const int             dimmx)
{
    for(int z = blockIdx.x * blockDim.x + threadIdx.x + nz0; 
            z < nzf; 
            z += gridDim.x * blockDim.x)
    {
        for(int x = blockIdx.y * blockDim.y + threadIdx.y + nx0; 
                x < nxf; 
                x += gridDim.y * blockDim.y)
        {
            for(int y = ny0; 
                    y < nyf; 
                    y++)
            {
                const float lrho = rho_BL(rho, z, x, y, dimmz, dimmx);
                
                const float stx  = stencil_X( SX, sxptr, dxi, z, x, y, dimmz, dimmx);
                const float sty  = stencil_Y( SY, syptr, dyi, z, x, y, dimmz, dimmx);
                const float stz  = stencil_Z( SZ, szptr, dzi, z, x, y, dimmz, dimmx);
                
                vptr[IDX(z,x,y,dimmz,dimmx)] += (stx  + sty  + stz) * dt * lrho;
            }
        }
    }
}
#endif

extern "C"
void compute_component_vcell_BL_cuda ( float* vptr,
                                 const float* szptr,
                                 const float* sxptr,
                                 const float* syptr,
                                 const float* rho,
                                 const float  dt,
                                 const float  dzi,
                                 const float  dxi,
                                 const float  dyi,
                                 const int    nz0,
                                 const int    nzf,
                                 const int    nx0,
                                 const int    nxf,
                                 const int    ny0,
                                 const int    nyf,
                                 const int    SZ,
                                 const int    SX,
                                 const int    SY,
                                 const int    dimmz,
                                 const int    dimmx,
                                 void*        stream)
{
    const int block_dim_x = 32;
    const int block_dim_y = 4;


    dim3 grid_dim( ((nzf-nz0) + block_dim_x-1)/block_dim_x,
                   ((nxf-nx0) + block_dim_y-1)/block_dim_y,
                      1 );
    dim3 block_dim(block_dim_x, block_dim_y, 1);

    hipStream_t s = (hipStream_t) stream;

#ifdef OPTIMIZED
    compute_component_vcell_BL_cuda_k<4,block_dim_x,block_dim_y><<<grid_dim, block_dim, 0, s>>>
        (vptr, szptr, sxptr, syptr, rho, dt, dzi, dxi, dyi, 
         nz0, nzf, nx0, nxf, ny0, nyf, SZ, SX, SY, dimmz, dimmx);
#else
    compute_component_vcell_BL_cuda_k<<<grid_dim, block_dim, 0, s>>>
        (vptr, szptr, sxptr, syptr, rho, dt, dzi, dxi, dyi, 
         nz0, nzf, nx0, nxf, ny0, nyf, SZ, SX, SY, dimmz, dimmx);
#endif
};





/* ------------------------------------------------------------------------------ */
/*                                                                                */
/*                               CALCULO DE TENSIONES                             */
/*                                                                                */
/* ------------------------------------------------------------------------------ */

__device__ inline
void stress_update(float* __restrict__ sptr,
                   const float     c1,
                   const float     c2,
                   const float     c3,
                   const float     c4,
                   const float     c5,
                   const float     c6,
                   const int  z,
                   const int  x,
                   const int  y,
                   const float     dt,
                   const float     u_x,
                   const float     u_y,
                   const float     u_z,
                   const float     v_x,
                   const float     v_y,
                   const float     v_z,
                   const float     w_x,
                   const float     w_y,
                   const float     w_z,
                   const int  dimmz,
                   const int  dimmx)
{
    float accum  = dt * c1 * u_x;
         accum += dt * c2 * v_y;
         accum += dt * c3 * w_z;
         accum += dt * c4 * (w_y + v_z);
         accum += dt * c5 * (w_x + u_z);
         accum += dt * c6 * (v_x + u_y);
    sptr[IDX(z,x,y,dimmz,dimmx)] += accum;
};

__device__ inline
float cell_coeff_BR (const float* __restrict__ ptr, 
                     const int z,
                     const int x,
                     const int y,
                     const int dimmz,
                     const int dimmx)
{
    return ( 1.0f / ( 2.5f  *(ptr[IDX(z  , x  ,y,dimmz,dimmx)] +
                              ptr[IDX(z  , x+1,y,dimmz,dimmx)] +
                              ptr[IDX(z+1, x  ,y,dimmz,dimmx)] +
                              ptr[IDX(z+1, x+1,y,dimmz,dimmx)])) );
};

__device__ inline
float cell_coeff_TL (const float* __restrict__ ptr, 
                     const int z, 
                     const int x, 
                     const int y, 
                     const int dimmz, 
                     const int dimmx)
{
    return ( 1.0f / (ptr[IDX(z,x,y,dimmz,dimmx)]));
};

__device__ inline
float cell_coeff_BL (const float* __restrict__ ptr, 
                     const int z, 
                     const int x, 
                     const int y, 
                     const int dimmz, 
                     const int dimmx)
{
    return ( 1.0f / ( 2.5f *(ptr[IDX(z  ,x,y  ,dimmz,dimmx)] +
                             ptr[IDX(z  ,x,y+1,dimmz,dimmx)] +
                             ptr[IDX(z+1,x,y  ,dimmz,dimmx)] +
                             ptr[IDX(z+1,x,y+1,dimmz,dimmx)])) );
};

__device__ inline
float cell_coeff_TR (const float* __restrict__ ptr, 
                     const int z, 
                     const int x, 
                     const int y, 
                     const int dimmz, 
                     const int dimmx)
{
    return ( 1.0f / ( 2.5f *(ptr[IDX(z  , x  , y  ,dimmz,dimmx)] +
                             ptr[IDX(z  , x+1, y  ,dimmz,dimmx)] +
                             ptr[IDX(z  , x  , y+1,dimmz,dimmx)] +
                             ptr[IDX(z  , x+1, y+1,dimmz,dimmx)])));
};

__device__ inline
float cell_coeff_ARTM_BR(const float* __restrict__ ptr, 
                         const int z, 
                         const int x, 
                         const int y, 
                         const int dimmz, 
                         const int dimmx)
{
    return ((1.0f / ptr[IDX(z  ,x  ,y,dimmz,dimmx )]  +
             1.0f / ptr[IDX(z  ,x+1,y,dimmz,dimmx )]  +
             1.0f / ptr[IDX(z+1,x  ,y,dimmz,dimmx )]  +
             1.0f / ptr[IDX(z+1,x+1,y,dimmz,dimmx )]) * 0.25f);
};

__device__ inline
float cell_coeff_ARTM_TL( const float* __restrict__ ptr, 
                         const int z, 
                         const int x, 
                         const int y, 
                         const int dimmz, 
                         const int dimmx)
{
    return (1.0f / ptr[IDX(z,x,y,dimmz,dimmx)]);
};

__device__ inline
float cell_coeff_ARTM_BL(const float* __restrict__ ptr, 
                         const int z, 
                         const int x, 
                         const int y, 
                         const int dimmz, 
                         const int dimmx)
{
    return ((1.0f / ptr[IDX(z  ,x,y  ,dimmz,dimmx)]  +
             1.0f / ptr[IDX(z  ,x,y+1,dimmz,dimmx)]  +
             1.0f / ptr[IDX(z+1,x,y  ,dimmz,dimmx)]  +
             1.0f / ptr[IDX(z+1,x,y+1,dimmz,dimmx)]) * 0.25f);
};

__device__ inline
float cell_coeff_ARTM_TR(const float* __restrict__ ptr, 
                         const int z, 
                         const int x, 
                         const int y, 
                         const int dimmz, 
                         const int dimmx)
{
    return ((1.0f / ptr[IDX(z,x  ,y  ,dimmz,dimmx)]  +
             1.0f / ptr[IDX(z,x+1,y  ,dimmz,dimmx)]  +
             1.0f / ptr[IDX(z,x  ,y+1,dimmz,dimmx)]  +
             1.0f / ptr[IDX(z,x+1,y+1,dimmz,dimmx)]) * 0.25f);
};



__global__
__launch_bounds__(128, 8) 
void compute_component_scell_TR_cuda_k ( float* __restrict__ sxxptr,
                                         float* __restrict__ syyptr,
                                         float* __restrict__ szzptr,
                                         float* __restrict__ syzptr,
                                         float* __restrict__ sxzptr,
                                         float* __restrict__ sxyptr,
                                   const float* __restrict__ vxu,
                                   const float* __restrict__ vxv,
                                   const float* __restrict__ vxw,
                                   const float* __restrict__ vyu,
                                   const float* __restrict__ vyv,
                                   const float* __restrict__ vyw,
                                   const float* __restrict__ vzu,
                                   const float* __restrict__ vzv,
                                   const float* __restrict__ vzw,
                                   const float* __restrict__ cc11,
                                   const float* __restrict__ cc12,
                                   const float* __restrict__ cc13,
                                   const float* __restrict__ cc14,
                                   const float* __restrict__ cc15,
                                   const float* __restrict__ cc16,
                                   const float* __restrict__ cc22,
                                   const float* __restrict__ cc23,
                                   const float* __restrict__ cc24,
                                   const float* __restrict__ cc25,
                                   const float* __restrict__ cc26,
                                   const float* __restrict__ cc33,
                                   const float* __restrict__ cc34,
                                   const float* __restrict__ cc35,
                                   const float* __restrict__ cc36,
                                   const float* __restrict__ cc44,
                                   const float* __restrict__ cc45,
                                   const float* __restrict__ cc46,
                                   const float* __restrict__ cc55,
                                   const float* __restrict__ cc56,
                                   const float* __restrict__ cc66,
                                   const float  dt,
                                   const float  dzi,
                                   const float  dxi,
                                   const float  dyi,
                                   const int    nz0,
                                   const int    nzf,
                                   const int    nx0,
                                   const int    nxf,
                                   const int    ny0,
                                   const int    nyf,
                                   const int    SZ,
                                   const int    SX,
                                   const int    SY,
                                   const int    dimmz,
                                   const int    dimmx)
{
    for(int z = blockIdx.x * blockDim.x + threadIdx.x + nz0; 
            z < nzf; 
            z += gridDim.x * blockDim.x)
    {
        for(int x = blockIdx.y * blockDim.y + threadIdx.y + nx0; 
                x < nxf; 
                x += gridDim.y * blockDim.y)
        {
            for(int y = ny0; 
                    y < nyf; 
                    y++)
            {
                const float c11 = cell_coeff_TR      (cc11, z, x, y, dimmz, dimmx);
                const float c12 = cell_coeff_TR      (cc12, z, x, y, dimmz, dimmx);
                const float c13 = cell_coeff_TR      (cc13, z, x, y, dimmz, dimmx);
                const float c14 = cell_coeff_ARTM_TR (cc14, z, x, y, dimmz, dimmx);
                const float c15 = cell_coeff_ARTM_TR (cc15, z, x, y, dimmz, dimmx);
                const float c16 = cell_coeff_ARTM_TR (cc16, z, x, y, dimmz, dimmx);
                const float c22 = cell_coeff_TR      (cc22, z, x, y, dimmz, dimmx);
                const float c23 = cell_coeff_TR      (cc23, z, x, y, dimmz, dimmx);
                const float c24 = cell_coeff_ARTM_TR (cc24, z, x, y, dimmz, dimmx);
                const float c25 = cell_coeff_ARTM_TR (cc25, z, x, y, dimmz, dimmx);
                const float c26 = cell_coeff_ARTM_TR (cc26, z, x, y, dimmz, dimmx);
                const float c33 = cell_coeff_TR      (cc33, z, x, y, dimmz, dimmx);
                const float c34 = cell_coeff_ARTM_TR (cc34, z, x, y, dimmz, dimmx);
                const float c35 = cell_coeff_ARTM_TR (cc35, z, x, y, dimmz, dimmx);
                const float c36 = cell_coeff_ARTM_TR (cc36, z, x, y, dimmz, dimmx);
                const float c44 = cell_coeff_TR      (cc44, z, x, y, dimmz, dimmx);
                const float c45 = cell_coeff_ARTM_TR (cc45, z, x, y, dimmz, dimmx);
                const float c46 = cell_coeff_ARTM_TR (cc46, z, x, y, dimmz, dimmx);
                const float c55 = cell_coeff_TR      (cc55, z, x, y, dimmz, dimmx);
                const float c56 = cell_coeff_ARTM_TR (cc56, z, x, y, dimmz, dimmx);
                const float c66 = cell_coeff_TR      (cc66, z, x, y, dimmz, dimmx);
                
                const float u_x = stencil_X (SX, vxu, dxi, z, x, y, dimmz, dimmx);
                const float v_x = stencil_X (SX, vxv, dxi, z, x, y, dimmz, dimmx);
                const float w_x = stencil_X (SX, vxw, dxi, z, x, y, dimmz, dimmx);
                
                const float u_y = stencil_Y (SY, vyu, dyi, z, x, y, dimmz, dimmx);
                const float v_y = stencil_Y (SY, vyv, dyi, z, x, y, dimmz, dimmx);
                const float w_y = stencil_Y (SY, vyw, dyi, z, x, y, dimmz, dimmx);
                
                const float u_z = stencil_Z (SZ, vzu, dzi, z, x, y, dimmz, dimmx);
                const float v_z = stencil_Z (SZ, vzv, dzi, z, x, y, dimmz, dimmx);
                const float w_z = stencil_Z (SZ, vzw, dzi, z, x, y, dimmz, dimmx);
                
                stress_update (sxxptr,c11,c12,c13,c14,c15,c16,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
                stress_update (syyptr,c12,c22,c23,c24,c25,c26,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
                stress_update (szzptr,c13,c23,c33,c34,c35,c36,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
                stress_update (syzptr,c14,c24,c34,c44,c45,c46,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
                stress_update (sxzptr,c15,c25,c35,c45,c55,c56,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
                stress_update (sxyptr,c16,c26,c36,c46,c56,c66,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
            }
        }
    }
}

extern "C"
void compute_component_scell_TR_cuda ( float* sxxptr,
                                       float* syyptr,
                                       float* szzptr,
                                       float* syzptr,
                                       float* sxzptr,
                                       float* sxyptr,
                                 const float* vxu,
                                 const float* vxv,
                                 const float* vxw,
                                 const float* vyu,
                                 const float* vyv,
                                 const float* vyw,
                                 const float* vzu,
                                 const float* vzv,
                                 const float* vzw,
                                 const float* cc11,
                                 const float* cc12,
                                 const float* cc13,
                                 const float* cc14,
                                 const float* cc15,
                                 const float* cc16,
                                 const float* cc22,
                                 const float* cc23,
                                 const float* cc24,
                                 const float* cc25,
                                 const float* cc26,
                                 const float* cc33,
                                 const float* cc34,
                                 const float* cc35,
                                 const float* cc36,
                                 const float* cc44,
                                 const float* cc45,
                                 const float* cc46,
                                 const float* cc55,
                                 const float* cc56,
                                 const float* cc66,
                                 const float  dt,
                                 const float  dzi,
                                 const float  dxi,
                                 const float  dyi,
                                 const int    nz0,
                                 const int    nzf,
                                 const int    nx0,
                                 const int    nxf,
                                 const int    ny0,
                                 const int    nyf,
                                 const int    SZ,
                                 const int    SX,
                                 const int    SY,
                                 const int    dimmz,
                                 const int    dimmx,
                                 void*        stream)
{
    const int block_dim_x = 32;
    const int block_dim_y = 4;


    dim3 grid_dim( ((nzf-nz0) + block_dim_x-1)/block_dim_x,
                   ((nxf-nx0) + block_dim_y-1)/block_dim_y,
                      1 );
    dim3 block_dim(block_dim_x, block_dim_y, 1);

    hipStream_t s = (hipStream_t) stream;

    compute_component_scell_TR_cuda_k<<<grid_dim, block_dim, 0, s>>>
        (sxxptr, syyptr, szzptr, syzptr, sxzptr, sxyptr,
         vxu, vxv, vxw, vyu, vyv, vyw, vzu, vzv, vzw,
         cc11, cc12, cc13, cc14, cc15, cc16,
         cc22, cc23, cc24, cc25, cc26,
         cc33, cc34, cc35, cc36,
         cc44, cc45, cc45,
         cc55, cc56,
         cc66,
         dt, dzi, dxi, dyi, nz0, nzf, nx0, nxf, ny0, nyf, SZ, SX, SY, dimmz, dimmx);
};


__global__
__launch_bounds__(128, 8) 
void compute_component_scell_TL_cuda_k ( float* __restrict__ sxxptr,
                                         float* __restrict__ syyptr,
                                         float* __restrict__ szzptr,
                                         float* __restrict__ syzptr,
                                         float* __restrict__ sxzptr,
                                         float* __restrict__ sxyptr,
                                   const float* __restrict__ vxu,
                                   const float* __restrict__ vxv,
                                   const float* __restrict__ vxw,
                                   const float* __restrict__ vyu,
                                   const float* __restrict__ vyv,
                                   const float* __restrict__ vyw,
                                   const float* __restrict__ vzu,
                                   const float* __restrict__ vzv,
                                   const float* __restrict__ vzw,
                                   const float* __restrict__ cc11,
                                   const float* __restrict__ cc12,
                                   const float* __restrict__ cc13,
                                   const float* __restrict__ cc14,
                                   const float* __restrict__ cc15,
                                   const float* __restrict__ cc16,
                                   const float* __restrict__ cc22,
                                   const float* __restrict__ cc23,
                                   const float* __restrict__ cc24,
                                   const float* __restrict__ cc25,
                                   const float* __restrict__ cc26,
                                   const float* __restrict__ cc33,
                                   const float* __restrict__ cc34,
                                   const float* __restrict__ cc35,
                                   const float* __restrict__ cc36,
                                   const float* __restrict__ cc44,
                                   const float* __restrict__ cc45,
                                   const float* __restrict__ cc46,
                                   const float* __restrict__ cc55,
                                   const float* __restrict__ cc56,
                                   const float* __restrict__ cc66,
                                   const float  dt,
                                   const float  dzi,
                                   const float  dxi,
                                   const float  dyi,
                                   const int    nz0,
                                   const int    nzf,
                                   const int    nx0,
                                   const int    nxf,
                                   const int    ny0,
                                   const int    nyf,
                                   const int    SZ,
                                   const int    SX,
                                   const int    SY,
                                   const int    dimmz,
                                   const int    dimmx)
{
    for(int z = blockIdx.x * blockDim.x + threadIdx.x + nz0; 
            z < nzf; 
            z += gridDim.x * blockDim.x)
    {
        for(int x = blockIdx.y * blockDim.y + threadIdx.y + nx0; 
                x < nxf; 
                x += gridDim.y * blockDim.y)
        {
            for(int y = ny0; 
                    y < nyf; 
                    y++)
            {
                const float c11 = cell_coeff_TL      (cc11, z, x, y, dimmz, dimmx);
                const float c12 = cell_coeff_TL      (cc12, z, x, y, dimmz, dimmx);
                const float c13 = cell_coeff_TL      (cc13, z, x, y, dimmz, dimmx);
                const float c14 = cell_coeff_ARTM_TL (cc14, z, x, y, dimmz, dimmx);
                const float c15 = cell_coeff_ARTM_TL (cc15, z, x, y, dimmz, dimmx);
                const float c16 = cell_coeff_ARTM_TL (cc16, z, x, y, dimmz, dimmx);
                const float c22 = cell_coeff_TL      (cc22, z, x, y, dimmz, dimmx);
                const float c23 = cell_coeff_TL      (cc23, z, x, y, dimmz, dimmx);
                const float c24 = cell_coeff_ARTM_TL (cc24, z, x, y, dimmz, dimmx);
                const float c25 = cell_coeff_ARTM_TL (cc25, z, x, y, dimmz, dimmx);
                const float c26 = cell_coeff_ARTM_TL (cc26, z, x, y, dimmz, dimmx);
                const float c33 = cell_coeff_TL      (cc33, z, x, y, dimmz, dimmx);
                const float c34 = cell_coeff_ARTM_TL (cc34, z, x, y, dimmz, dimmx);
                const float c35 = cell_coeff_ARTM_TL (cc35, z, x, y, dimmz, dimmx);
                const float c36 = cell_coeff_ARTM_TL (cc36, z, x, y, dimmz, dimmx);
                const float c44 = cell_coeff_TL      (cc44, z, x, y, dimmz, dimmx);
                const float c45 = cell_coeff_ARTM_TL (cc45, z, x, y, dimmz, dimmx);
                const float c46 = cell_coeff_ARTM_TL (cc46, z, x, y, dimmz, dimmx);
                const float c55 = cell_coeff_TL      (cc55, z, x, y, dimmz, dimmx);
                const float c56 = cell_coeff_ARTM_TL (cc56, z, x, y, dimmz, dimmx);
                const float c66 = cell_coeff_TL      (cc66, z, x, y, dimmz, dimmx);
                
                const float u_x = stencil_X (SX, vxu, dxi, z, x, y, dimmz, dimmx);
                const float v_x = stencil_X (SX, vxv, dxi, z, x, y, dimmz, dimmx);
                const float w_x = stencil_X (SX, vxw, dxi, z, x, y, dimmz, dimmx);
                
                const float u_y = stencil_Y (SY, vyu, dyi, z, x, y, dimmz, dimmx);
                const float v_y = stencil_Y (SY, vyv, dyi, z, x, y, dimmz, dimmx);
                const float w_y = stencil_Y (SY, vyw, dyi, z, x, y, dimmz, dimmx);
                
                const float u_z = stencil_Z (SZ, vzu, dzi, z, x, y, dimmz, dimmx);
                const float v_z = stencil_Z (SZ, vzv, dzi, z, x, y, dimmz, dimmx);
                const float w_z = stencil_Z (SZ, vzw, dzi, z, x, y, dimmz, dimmx);
                
                stress_update (sxxptr,c11,c12,c13,c14,c15,c16,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
                stress_update (syyptr,c12,c22,c23,c24,c25,c26,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
                stress_update (szzptr,c13,c23,c33,c34,c35,c36,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
                stress_update (syzptr,c14,c24,c34,c44,c45,c46,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
                stress_update (sxzptr,c15,c25,c35,c45,c55,c56,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
                stress_update (sxyptr,c16,c26,c36,c46,c56,c66,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
            }
        }
    }
}

extern "C"
void compute_component_scell_TL_cuda ( float* sxxptr,
                                       float* syyptr,
                                       float* szzptr,
                                       float* syzptr,
                                       float* sxzptr,
                                       float* sxyptr,
                                 const float* vxu,
                                 const float* vxv,
                                 const float* vxw,
                                 const float* vyu,
                                 const float* vyv,
                                 const float* vyw,
                                 const float* vzu,
                                 const float* vzv,
                                 const float* vzw,
                                 const float* cc11,
                                 const float* cc12,
                                 const float* cc13,
                                 const float* cc14,
                                 const float* cc15,
                                 const float* cc16,
                                 const float* cc22,
                                 const float* cc23,
                                 const float* cc24,
                                 const float* cc25,
                                 const float* cc26,
                                 const float* cc33,
                                 const float* cc34,
                                 const float* cc35,
                                 const float* cc36,
                                 const float* cc44,
                                 const float* cc45,
                                 const float* cc46,
                                 const float* cc55,
                                 const float* cc56,
                                 const float* cc66,
                                 const float  dt,
                                 const float  dzi,
                                 const float  dxi,
                                 const float  dyi,
                                 const int    nz0,
                                 const int    nzf,
                                 const int    nx0,
                                 const int    nxf,
                                 const int    ny0,
                                 const int    nyf,
                                 const int    SZ,
                                 const int    SX,
                                 const int    SY,
                                 const int    dimmz,
                                 const int    dimmx,
                                 void*        stream)
{
    const int block_dim_x = 32;
    const int block_dim_y = 4;


    dim3 grid_dim( ((nzf-nz0) + block_dim_x-1)/block_dim_x,
                   ((nxf-nx0) + block_dim_y-1)/block_dim_y,
                      1 );
    dim3 block_dim(block_dim_x, block_dim_y, 1);

    hipStream_t s = (hipStream_t) stream;

    compute_component_scell_TL_cuda_k<<<grid_dim, block_dim, 0, s>>>
        (sxxptr, syyptr, szzptr, syzptr, sxzptr, sxyptr,
         vxu, vxv, vxw, vyu, vyv, vyw, vzu, vzv, vzw,
         cc11, cc12, cc13, cc14, cc15, cc16,
         cc22, cc23, cc24, cc25, cc26,
         cc33, cc34, cc35, cc36,
         cc44, cc45, cc45,
         cc55, cc56,
         cc66,
         dt, dzi, dxi, dyi, nz0, nzf, nx0, nxf, ny0, nyf, SZ, SX, SY, dimmz, dimmx);
};


__global__
__launch_bounds__(128, 8) 
void compute_component_scell_BR_cuda_k ( float* __restrict__ sxxptr,
                                         float* __restrict__ syyptr,
                                         float* __restrict__ szzptr,
                                         float* __restrict__ syzptr,
                                         float* __restrict__ sxzptr,
                                         float* __restrict__ sxyptr,
                                   const float* __restrict__ vxu,
                                   const float* __restrict__ vxv,
                                   const float* __restrict__ vxw,
                                   const float* __restrict__ vyu,
                                   const float* __restrict__ vyv,
                                   const float* __restrict__ vyw,
                                   const float* __restrict__ vzu,
                                   const float* __restrict__ vzv,
                                   const float* __restrict__ vzw,
                                   const float* __restrict__ cc11,
                                   const float* __restrict__ cc12,
                                   const float* __restrict__ cc13,
                                   const float* __restrict__ cc14,
                                   const float* __restrict__ cc15,
                                   const float* __restrict__ cc16,
                                   const float* __restrict__ cc22,
                                   const float* __restrict__ cc23,
                                   const float* __restrict__ cc24,
                                   const float* __restrict__ cc25,
                                   const float* __restrict__ cc26,
                                   const float* __restrict__ cc33,
                                   const float* __restrict__ cc34,
                                   const float* __restrict__ cc35,
                                   const float* __restrict__ cc36,
                                   const float* __restrict__ cc44,
                                   const float* __restrict__ cc45,
                                   const float* __restrict__ cc46,
                                   const float* __restrict__ cc55,
                                   const float* __restrict__ cc56,
                                   const float* __restrict__ cc66,
                                   const float  dt,
                                   const float  dzi,
                                   const float  dxi,
                                   const float  dyi,
                                   const int    nz0,
                                   const int    nzf,
                                   const int    nx0,
                                   const int    nxf,
                                   const int    ny0,
                                   const int    nyf,
                                   const int    SZ,
                                   const int    SX,
                                   const int    SY,
                                   const int    dimmz,
                                   const int    dimmx)
{
    for(int z = blockIdx.x * blockDim.x + threadIdx.x + nz0; 
            z < nzf; 
            z += gridDim.x * blockDim.x)
    {
        for(int x = blockIdx.y * blockDim.y + threadIdx.y + nx0; 
                x < nxf; 
                x += gridDim.y * blockDim.y)
        {
            for(int y = ny0; 
                    y < nyf; 
                    y++)
            {
                const float c11 = cell_coeff_BR      (cc11, z, x, y, dimmz, dimmx);
                const float c12 = cell_coeff_BR      (cc12, z, x, y, dimmz, dimmx);
                const float c13 = cell_coeff_BR      (cc13, z, x, y, dimmz, dimmx);
                const float c14 = cell_coeff_ARTM_BR (cc14, z, x, y, dimmz, dimmx);
                const float c15 = cell_coeff_ARTM_BR (cc15, z, x, y, dimmz, dimmx);
                const float c16 = cell_coeff_ARTM_BR (cc16, z, x, y, dimmz, dimmx);
                const float c22 = cell_coeff_BR      (cc22, z, x, y, dimmz, dimmx);
                const float c23 = cell_coeff_BR      (cc23, z, x, y, dimmz, dimmx);
                const float c24 = cell_coeff_ARTM_BR (cc24, z, x, y, dimmz, dimmx);
                const float c25 = cell_coeff_ARTM_BR (cc25, z, x, y, dimmz, dimmx);
                const float c26 = cell_coeff_ARTM_BR (cc26, z, x, y, dimmz, dimmx);
                const float c33 = cell_coeff_BR      (cc33, z, x, y, dimmz, dimmx);
                const float c34 = cell_coeff_ARTM_BR (cc34, z, x, y, dimmz, dimmx);
                const float c35 = cell_coeff_ARTM_BR (cc35, z, x, y, dimmz, dimmx);
                const float c36 = cell_coeff_ARTM_BR (cc36, z, x, y, dimmz, dimmx);
                const float c44 = cell_coeff_BR      (cc44, z, x, y, dimmz, dimmx);
                const float c45 = cell_coeff_ARTM_BR (cc45, z, x, y, dimmz, dimmx);
                const float c46 = cell_coeff_ARTM_BR (cc46, z, x, y, dimmz, dimmx);
                const float c55 = cell_coeff_BR      (cc55, z, x, y, dimmz, dimmx);
                const float c56 = cell_coeff_ARTM_BR (cc56, z, x, y, dimmz, dimmx);
                const float c66 = cell_coeff_BR      (cc66, z, x, y, dimmz, dimmx);
                
                const float u_x = stencil_X (SX, vxu, dxi, z, x, y, dimmz, dimmx);
                const float v_x = stencil_X (SX, vxv, dxi, z, x, y, dimmz, dimmx);
                const float w_x = stencil_X (SX, vxw, dxi, z, x, y, dimmz, dimmx);
                
                const float u_y = stencil_Y (SY, vyu, dyi, z, x, y, dimmz, dimmx);
                const float v_y = stencil_Y (SY, vyv, dyi, z, x, y, dimmz, dimmx);
                const float w_y = stencil_Y (SY, vyw, dyi, z, x, y, dimmz, dimmx);
                
                const float u_z = stencil_Z (SZ, vzu, dzi, z, x, y, dimmz, dimmx);
                const float v_z = stencil_Z (SZ, vzv, dzi, z, x, y, dimmz, dimmx);
                const float w_z = stencil_Z (SZ, vzw, dzi, z, x, y, dimmz, dimmx);
                
                stress_update (sxxptr,c11,c12,c13,c14,c15,c16,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
                stress_update (syyptr,c12,c22,c23,c24,c25,c26,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
                stress_update (szzptr,c13,c23,c33,c34,c35,c36,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
                stress_update (syzptr,c14,c24,c34,c44,c45,c46,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
                stress_update (sxzptr,c15,c25,c35,c45,c55,c56,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
                stress_update (sxyptr,c16,c26,c36,c46,c56,c66,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
            }
        }
    }
}

extern "C"
void compute_component_scell_BR_cuda ( float* sxxptr,
                                       float* syyptr,
                                       float* szzptr,
                                       float* syzptr,
                                       float* sxzptr,
                                       float* sxyptr,
                                 const float* vxu,
                                 const float* vxv,
                                 const float* vxw,
                                 const float* vyu,
                                 const float* vyv,
                                 const float* vyw,
                                 const float* vzu,
                                 const float* vzv,
                                 const float* vzw,
                                 const float* cc11,
                                 const float* cc12,
                                 const float* cc13,
                                 const float* cc14,
                                 const float* cc15,
                                 const float* cc16,
                                 const float* cc22,
                                 const float* cc23,
                                 const float* cc24,
                                 const float* cc25,
                                 const float* cc26,
                                 const float* cc33,
                                 const float* cc34,
                                 const float* cc35,
                                 const float* cc36,
                                 const float* cc44,
                                 const float* cc45,
                                 const float* cc46,
                                 const float* cc55,
                                 const float* cc56,
                                 const float* cc66,
                                 const float  dt,
                                 const float  dzi,
                                 const float  dxi,
                                 const float  dyi,
                                 const int    nz0,
                                 const int    nzf,
                                 const int    nx0,
                                 const int    nxf,
                                 const int    ny0,
                                 const int    nyf,
                                 const int    SZ,
                                 const int    SX,
                                 const int    SY,
                                 const int    dimmz,
                                 const int    dimmx,
                                 void*        stream)
{
    const int block_dim_x = 32;
    const int block_dim_y = 4;


    dim3 grid_dim( ((nzf-nz0) + block_dim_x-1)/block_dim_x,
                   ((nxf-nx0) + block_dim_y-1)/block_dim_y,
                      1 );
    dim3 block_dim(block_dim_x, block_dim_y, 1);

    hipStream_t s = (hipStream_t) stream;

    compute_component_scell_BR_cuda_k<<<grid_dim, block_dim, 0, s>>>
        (sxxptr, syyptr, szzptr, syzptr, sxzptr, sxyptr,
         vxu, vxv, vxw, vyu, vyv, vyw, vzu, vzv, vzw,
         cc11, cc12, cc13, cc14, cc15, cc16,
         cc22, cc23, cc24, cc25, cc26,
         cc33, cc34, cc35, cc36,
         cc44, cc45, cc45,
         cc55, cc56,
         cc66,
         dt, dzi, dxi, dyi, nz0, nzf, nx0, nxf, ny0, nyf, SZ, SX, SY, dimmz, dimmx);
};


__global__
__launch_bounds__(128, 8) 
void compute_component_scell_BL_cuda_k ( float* __restrict__ sxxptr,
                                         float* __restrict__ syyptr,
                                         float* __restrict__ szzptr,
                                         float* __restrict__ syzptr,
                                         float* __restrict__ sxzptr,
                                         float* __restrict__ sxyptr,
                                   const float* __restrict__ vxu,
                                   const float* __restrict__ vxv,
                                   const float* __restrict__ vxw,
                                   const float* __restrict__ vyu,
                                   const float* __restrict__ vyv,
                                   const float* __restrict__ vyw,
                                   const float* __restrict__ vzu,
                                   const float* __restrict__ vzv,
                                   const float* __restrict__ vzw,
                                   const float* __restrict__ cc11,
                                   const float* __restrict__ cc12,
                                   const float* __restrict__ cc13,
                                   const float* __restrict__ cc14,
                                   const float* __restrict__ cc15,
                                   const float* __restrict__ cc16,
                                   const float* __restrict__ cc22,
                                   const float* __restrict__ cc23,
                                   const float* __restrict__ cc24,
                                   const float* __restrict__ cc25,
                                   const float* __restrict__ cc26,
                                   const float* __restrict__ cc33,
                                   const float* __restrict__ cc34,
                                   const float* __restrict__ cc35,
                                   const float* __restrict__ cc36,
                                   const float* __restrict__ cc44,
                                   const float* __restrict__ cc45,
                                   const float* __restrict__ cc46,
                                   const float* __restrict__ cc55,
                                   const float* __restrict__ cc56,
                                   const float* __restrict__ cc66,
                                   const float  dt,
                                   const float  dzi,
                                   const float  dxi,
                                   const float  dyi,
                                   const int    nz0,
                                   const int    nzf,
                                   const int    nx0,
                                   const int    nxf,
                                   const int    ny0,
                                   const int    nyf,
                                   const int    SZ,
                                   const int    SX,
                                   const int    SY,
                                   const int    dimmz,
                                   const int    dimmx)
{
    for(int z = blockIdx.x * blockDim.x + threadIdx.x + nz0; 
            z < nzf; 
            z += gridDim.x * blockDim.x)
    {
        for(int x = blockIdx.y * blockDim.y + threadIdx.y + nx0; 
                x < nxf; 
                x += gridDim.y * blockDim.y)
        {
            for(int y = ny0; 
                    y < nyf; 
                    y++)
            {
                const float c11 = cell_coeff_BL      (cc11, z, x, y, dimmz, dimmx);
                const float c12 = cell_coeff_BL      (cc12, z, x, y, dimmz, dimmx);
                const float c13 = cell_coeff_BL      (cc13, z, x, y, dimmz, dimmx);
                const float c14 = cell_coeff_ARTM_BL (cc14, z, x, y, dimmz, dimmx);
                const float c15 = cell_coeff_ARTM_BL (cc15, z, x, y, dimmz, dimmx);
                const float c16 = cell_coeff_ARTM_BL (cc16, z, x, y, dimmz, dimmx);
                const float c22 = cell_coeff_BL      (cc22, z, x, y, dimmz, dimmx);
                const float c23 = cell_coeff_BL      (cc23, z, x, y, dimmz, dimmx);
                const float c24 = cell_coeff_ARTM_BL (cc24, z, x, y, dimmz, dimmx);
                const float c25 = cell_coeff_ARTM_BL (cc25, z, x, y, dimmz, dimmx);
                const float c26 = cell_coeff_ARTM_BL (cc26, z, x, y, dimmz, dimmx);
                const float c33 = cell_coeff_BL      (cc33, z, x, y, dimmz, dimmx);
                const float c34 = cell_coeff_ARTM_BL (cc34, z, x, y, dimmz, dimmx);
                const float c35 = cell_coeff_ARTM_BL (cc35, z, x, y, dimmz, dimmx);
                const float c36 = cell_coeff_ARTM_BL (cc36, z, x, y, dimmz, dimmx);
                const float c44 = cell_coeff_BL      (cc44, z, x, y, dimmz, dimmx);
                const float c45 = cell_coeff_ARTM_BL (cc45, z, x, y, dimmz, dimmx);
                const float c46 = cell_coeff_ARTM_BL (cc46, z, x, y, dimmz, dimmx);
                const float c55 = cell_coeff_BL      (cc55, z, x, y, dimmz, dimmx);
                const float c56 = cell_coeff_ARTM_BL (cc56, z, x, y, dimmz, dimmx);
                const float c66 = cell_coeff_BL      (cc66, z, x, y, dimmz, dimmx);
                
                const float u_x = stencil_X (SX, vxu, dxi, z, x, y, dimmz, dimmx);
                const float v_x = stencil_X (SX, vxv, dxi, z, x, y, dimmz, dimmx);
                const float w_x = stencil_X (SX, vxw, dxi, z, x, y, dimmz, dimmx);
                
                const float u_y = stencil_Y (SY, vyu, dyi, z, x, y, dimmz, dimmx);
                const float v_y = stencil_Y (SY, vyv, dyi, z, x, y, dimmz, dimmx);
                const float w_y = stencil_Y (SY, vyw, dyi, z, x, y, dimmz, dimmx);
                
                const float u_z = stencil_Z (SZ, vzu, dzi, z, x, y, dimmz, dimmx);
                const float v_z = stencil_Z (SZ, vzv, dzi, z, x, y, dimmz, dimmx);
                const float w_z = stencil_Z (SZ, vzw, dzi, z, x, y, dimmz, dimmx);
                
                stress_update (sxxptr,c11,c12,c13,c14,c15,c16,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
                stress_update (syyptr,c12,c22,c23,c24,c25,c26,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
                stress_update (szzptr,c13,c23,c33,c34,c35,c36,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
                stress_update (syzptr,c14,c24,c34,c44,c45,c46,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
                stress_update (sxzptr,c15,c25,c35,c45,c55,c56,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
                stress_update (sxyptr,c16,c26,c36,c46,c56,c66,z,x,y,dt,u_x,u_y,u_z,v_x,v_y,v_z,w_x,w_y,w_z,dimmz,dimmx );
            }
        }
    }
}

extern "C"
void compute_component_scell_BL_cuda ( float* sxxptr,
                                       float* syyptr,
                                       float* szzptr,
                                       float* syzptr,
                                       float* sxzptr,
                                       float* sxyptr,
                                 const float* vxu,
                                 const float* vxv,
                                 const float* vxw,
                                 const float* vyu,
                                 const float* vyv,
                                 const float* vyw,
                                 const float* vzu,
                                 const float* vzv,
                                 const float* vzw,
                                 const float* cc11,
                                 const float* cc12,
                                 const float* cc13,
                                 const float* cc14,
                                 const float* cc15,
                                 const float* cc16,
                                 const float* cc22,
                                 const float* cc23,
                                 const float* cc24,
                                 const float* cc25,
                                 const float* cc26,
                                 const float* cc33,
                                 const float* cc34,
                                 const float* cc35,
                                 const float* cc36,
                                 const float* cc44,
                                 const float* cc45,
                                 const float* cc46,
                                 const float* cc55,
                                 const float* cc56,
                                 const float* cc66,
                                 const float  dt,
                                 const float  dzi,
                                 const float  dxi,
                                 const float  dyi,
                                 const int    nz0,
                                 const int    nzf,
                                 const int    nx0,
                                 const int    nxf,
                                 const int    ny0,
                                 const int    nyf,
                                 const int    SZ,
                                 const int    SX,
                                 const int    SY,
                                 const int    dimmz,
                                 const int    dimmx,
                                 void*        stream)
{
    const int block_dim_x = 32;
    const int block_dim_y = 4;


    dim3 grid_dim( ((nzf-nz0) + block_dim_x-1)/block_dim_x,
                   ((nxf-nx0) + block_dim_y-1)/block_dim_y,
                      1 );
    dim3 block_dim(block_dim_x, block_dim_y, 1);

    hipStream_t s = (hipStream_t) stream;

    compute_component_scell_BL_cuda_k<<<grid_dim, block_dim, 0, s>>>
        (sxxptr, syyptr, szzptr, syzptr, sxzptr, sxyptr,
         vxu, vxv, vxw, vyu, vyv, vyw, vzu, vzv, vzw,
         cc11, cc12, cc13, cc14, cc15, cc16,
         cc22, cc23, cc24, cc25, cc26,
         cc33, cc34, cc35, cc36,
         cc44, cc45, cc45,
         cc55, cc56,
         cc66,
         dt, dzi, dxi, dyi, nz0, nzf, nx0, nxf, ny0, nyf, SZ, SX, SY, dimmz, dimmx);
};


